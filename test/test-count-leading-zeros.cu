#include "hip/hip_runtime.h"
#include <algorithm>
#include <variant>
#include <vector>

#include <Eigen/Sparse>

#include <gsl-lite/gsl-lite.hpp>

#include <boost/test/data/test_case.hpp>
#include <boost/test/unit_test.hpp>

#include <thrustshift/CSR.h>
#include <thrustshift/bit.h>
#include <thrustshift/defines.h>
#include <thrustshift/managed-vector.h>
#include <thrustshift/memory-resource.h>

namespace bdata = boost::unit_test::data;
using namespace thrustshift;

namespace kernel {
template <typename I>
__global__ void clz(I i, int* result) {

	*result = count_leading_zeros(i);
}

} // namespace kernel

BOOST_AUTO_TEST_CASE(test_count_leading_zeros) {

	managed_vector<int> device_result(1);

	{
		const int i = 0b00000000000000000000000000000000;
		const int gold_result = 32;
		kernel::clz<int><<<1, 1>>>(i, device_result.data());
		THRUSTSHIFT_CHECK_CUDA_ERROR(hipGetLastError());
		THRUSTSHIFT_CHECK_CUDA_ERROR(hipDeviceSynchronize());
		BOOST_TEST(device_result[0] == gold_result);
	}
	{
		const int i = 0b00000000000000000000000000000011;
		const int gold_result = 30;
		kernel::clz<int><<<1, 1>>>(i, device_result.data());
		THRUSTSHIFT_CHECK_CUDA_ERROR(hipGetLastError());
		THRUSTSHIFT_CHECK_CUDA_ERROR(hipDeviceSynchronize());
		BOOST_TEST(device_result[0] == gold_result);
	}
	{
		const long long int i =
		    0b0000000000000000000000000000001100000000000000000000000000000011;
		const long long int gold_result = 30;
		kernel::clz<long long int><<<1, 1>>>(i, device_result.data());

		THRUSTSHIFT_CHECK_CUDA_ERROR(hipGetLastError());
		THRUSTSHIFT_CHECK_CUDA_ERROR(hipDeviceSynchronize());
		BOOST_TEST(device_result[0] == gold_result);
	}

	{
		const long long int i =
		    0b0000000000000000000000000000000000000000000000000000000000000000;
		const long long int gold_result = 64;
		kernel::clz<long long int><<<1, 1>>>(i, device_result.data());
		THRUSTSHIFT_CHECK_CUDA_ERROR(hipGetLastError());
		THRUSTSHIFT_CHECK_CUDA_ERROR(hipDeviceSynchronize());
		BOOST_TEST(device_result[0] == gold_result);
	}
}
