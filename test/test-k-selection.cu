#include "hip/hip_runtime.h"
#include <algorithm>
#include <bitset>
#include <numeric>
#include <type_traits>
#include <variant>
#include <vector>

#include <Eigen/Sparse>

#include <gsl-lite/gsl-lite.hpp>

#include <cuda/runtime_api.hpp>

#include <boost/test/data/test_case.hpp>
#include <boost/test/unit_test.hpp>

#include <thrustshift/CSR.h>
#include <thrustshift/k-selection.h>
#include <thrustshift/managed-vector.h>
#include <thrustshift/memory-resource.h>
#include <thrustshift/random.h>
#include <thrustshift/sort.h>

namespace bdata = boost::unit_test::data;
using namespace thrustshift;

namespace {

template <typename Range>
auto k_largest_abs_values_gold(Range&& r, int k) {

	gsl_Expects(k >= 1);
	using T = typename std::remove_reference<Range>::type::value_type;
	const int N = r.size();
	std::vector<std::tuple<T, int>> v(N);
	for (int i = 0; i < N; ++i) {
		v[i] = std::tuple<T, int>{r[i], i};
	}
	std::sort(v.begin(), v.end(), [](auto a, auto b) {
		return std::abs(std::get<0>(a)) > std::abs(std::get<0>(b));
	});
	v.resize(k);
	return v;
}

template <typename T>
struct k_selection_test_data_t {
	std::vector<T> values;
	int k;
};

template <typename T>
std::ostream& operator<<(std::ostream& os,
                         const k_selection_test_data_t<T>& td) {
	os << "k = " << td.k << '\n';
	const size_t N = td.values.size();
	os << "values = [";
	if (N <= 100 && N > 0) {
		for (size_t i = 0; i < N - 1; ++i) {
			os << td.values[i] << ", ";
		}
		os << td.values[N - 1] << "]\n";
	}
	return os;
}

template<typename T>
struct uniform_distribution_type_proxy {
	using type = std::uniform_int_distribution<T>;
};
template<>
struct uniform_distribution_type_proxy<float> {
	using type = std::uniform_real_distribution<float>;
};
template<>
struct uniform_distribution_type_proxy<double> {
	using type = std::uniform_real_distribution<double>;
};

template <typename T>
auto gen_uniform_values(std::size_t N, T min, T max) {
	std::vector<T> v(N);
	std::default_random_engine rng;
	typename uniform_distribution_type_proxy<T>::type dist(min, max);
	// auto dist = [&] { // NOTE: produces still warnings. Wait for better compiler
	// 	if constexpr (std::is_integral<T>::value) {
	// 		return std::uniform_int_distribution<T>(min, max);
	// 	}
	// 	else {
	// 		return std::uniform_real_distribution<T>(min, max);
	// 	}
	// }();
	for (auto& e : v) {
		e = dist(rng);
	}
	return v;
}

std::vector<k_selection_test_data_t<int>> int_test_datas = {
    {gen_uniform_values<int>(100, 0, 765), 7},
    {gen_uniform_values<int>(100, 0, 765), 1},
    {gen_uniform_values<int>(100, 0, 765), 100},
    {gen_uniform_values<int>(100, 0, std::numeric_limits<int>::max()), 12},
    {gen_uniform_values<int>(100,
                                 std::numeric_limits<int>::min(),
                                 std::numeric_limits<int>::max()),
     13},
    {gen_uniform_values<int>(100, -6478, 765), 56},
    {gen_uniform_values<int>(476168, 0, 765), 10},
    {gen_uniform_values<int>(476168, -6884, 765), 5},
};

std::vector<k_selection_test_data_t<float>> float_test_datas = {
    {gen_uniform_values<float>(100, 0, 765), 7},
    {gen_uniform_values<float>(100, 0, 765), 1},
    {gen_uniform_values<float>(100, 0, 765), 100},
    {gen_uniform_values<float>(100, 0, std::numeric_limits<float>::max()), 12},
    {gen_uniform_values<float>(100,
                                 std::numeric_limits<float>::min(),
                                 std::numeric_limits<float>::max()),
     13},
    {gen_uniform_values<float>(100, -6478, 765), 56},
    {gen_uniform_values<float>(476168, 0, 765), 10},
    {gen_uniform_values<float>(476168, -6884, 765), 5},
};

} // namespace

template <typename T>
void do_k_selection_test(const k_selection_test_data_t<T>& td) {
	const thrustshift::managed_vector<T> v(td.values.begin(), td.values.end());
	const int k = td.k;
	const std::size_t N = v.size();

	auto device = cuda::device::current::get();
	auto stream = device.default_stream();

	const auto selected_values_gold = k_largest_abs_values_gold(v, k);

	thrustshift::pmr::delayed_pool_type<thrustshift::pmr::managed_resource_type>
	    delayed_memory_resource;

	thrustshift::managed_vector<thrust::tuple<T, int>> selected_values(N);

	select_k_largest_values_abs<T>(
	    stream, v, selected_values, k, delayed_memory_resource);

	std::set<T> gold_selected_unique;
	std::set<T> contender_selected_unique;
	for (const auto& e : selected_values_gold) {
		gold_selected_unique.insert(std::get<0>(e));
	}
	for (const auto& e : selected_values) {
		contender_selected_unique.insert(thrust::get<0>(e));
	}
	const bool p = gold_selected_unique == contender_selected_unique;
	for (const auto& g : gold_selected_unique) {
		BOOST_TEST_CONTEXT("gold value  = " << g) {
			const bool p = contender_selected_unique.find(g) !=
			               contender_selected_unique.end();
			BOOST_TEST(p);
		}
	}
}

BOOST_DATA_TEST_CASE(test_k_selection_int, int_test_datas, td) {
	do_k_selection_test(td);
}

BOOST_DATA_TEST_CASE(test_k_selection_float, float_test_datas, td) {
	do_k_selection_test(td);
}
